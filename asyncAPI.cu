// CPU与GPU同时执行


#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


int main()
{
  //show the device information
  int devID=0;
  hipDeviceProp_t deviceProps;
  checkCudaErrors(hipGetDeviceProperties(&deviceProps, devID));
  printf("CUDA device [%s]\n", deviceProps.name);

}

