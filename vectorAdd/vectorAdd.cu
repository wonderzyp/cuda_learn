//简单的向量加法：A+B=C


#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void vectorAdd(const float *A, const float *B, float *C,
                          int numElements) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i<numElements) {
    C[i] = A[i] + B[i] + 0.0f;
  }
}


int main()
{
  int numElements = 50000;

  size_t size = numElements * sizeof(float);

  float *h_A = (float *)malloc(size);
  float *h_B = (float *)malloc(size);
  float *h_C = (float *)malloc(size);

//分配内存后，需验证操作是否成功
  if (h_A == nullptr || h_B == nullptr || h_C==nullptr){
    fprintf(stderr, "Failed to allocate host vectors\n");
    exit(EXIT_FAILURE);
  }

  // Initialize the host input vectors
  for (int i = 0; i < numElements; ++i) {
    h_A[i] = rand() / (float)RAND_MAX;
    h_B[i] = rand() / (float)RAND_MAX;
  }

  float *d_A = nullptr;
  float *d_B = nullptr;
  float *d_C = nullptr;
  hipMalloc((void **)&d_A, size);
  hipMalloc((void **)&d_B, size);
  hipMalloc((void **)&d_C, size);

  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

  int threadsPerBlock = 256;
  int blocksPerGrid = (numElements+threadsPerBlock-1) / threadsPerBlock;
  printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid,
        threadsPerBlock);

  vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);

  hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);


  //验证GPU计算是否准确
  for (int i = 0; i < numElements; ++i) {
    if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5) {
      fprintf(stderr, "Result verification failed at element %d!\n", i);
      exit(EXIT_FAILURE);
    }
  }
  printf("Test PASSED\n");


  //释放资源
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  free(h_A);
  free(h_B);
  free(h_C);

  printf("Done\n");
  return 0;

}